#include "hip/hip_runtime.h"
/*cluster-wise parallel computation for expand phase*/
__global__ void expand_phase(int* join_list,cluster*cluster_list,int*cluster_track_vertex,int*neigh_conn_list,int no_node,int* added,int*now_added,edge_list*adj_list,int KCORE,int*expand,int maxi){

	__shared__ int expand1[1];
	
	int x=threadIdx.x,i,count=0,k,j;
	
	if(cluster_list[x].tag!=-1){
	    
		for(j=0;j<no_node;j++){//for each member of added_i
		    if(added[x*no_node+j]!=-1){
				for(k=0;k<no_node;k++){//for each member u_k of neigh of v_j
					if(neigh_conn_list[x*no_node + adj_list[j*no_node+k].vert]>join_list[k] && cluster_track_vertex[x*no_node+adj_list[j*no_node+k].vert]==-1)
							{
						cluster_track_vertex[x*no_node+adj_list[j*no_node+k].vert]=1;
						now_added[x*no_node+adj_list[j*no_node+k].vert]=1;
						count++;
			 		}
				}
			}
			
		
		}
		if(count>=1){
				expand1[0]=1;
				__syncthreads();
				*expand=expand1[0];
			}
	
  	}	
  
}
	
	
		
			
