#include "hip/hip_runtime.h"
/*to find the number of clusters a vertex belongs to...computed in parallel for all vertices*/
__global__ void count_cluster(int *c_neigh_list,edge_list *adj_list,cluster *cluster_list,int no_node)
{
			int x=threadIdx.x*no_node;
		
			int i=0,j,l,k,count1=0;
			
			for(i=0;i<no_node && cluster_list[threadIdx.x].tag!=-1;i++)
			{
					
			    int y=cluster_list[threadIdx.x].head;
			
				for(j=y;j<(y+no_node);j++)
				{
				    count1=0;
					if(adj_list[j].vert==i)
					{
						
						
						for(l=i*no_node+1;l<(i*no_node+no_node);l++)
					    {
					
					      for(k=y;k<(y+no_node);k++)
					      {
					         if(adj_list[l].vert!=-1 && adj_list[l].vert==adj_list[k].vert)
					         {
					         		count1++;
					         		break;
					         }
					         
					       }  
					      	
					    }
					    
						
						
					  c_neigh_list[x+i]=count1;										
					  break;
					}
					
			    }		
			    
			
			}
			//c_neigh_list[cluster_list[x].head]=x;
			/*int y=cluster_list[threadIdx.x].head;
			
			
			for(i=y;i<(y+no_node);i++)
			{
				if(adj_list[i]==threadIdx.x)
				{
					//vertex exist in the cluster
					count=0;
					for(j=x+1;j<(x+no_node);j++)
					{
					
					      for(k=y;k<(y+no_node);k++)
					         if(adj_list[j]==adj_list[k])
					         {
					         		count++;
					         }
					      	
					}
					c_neigh_list[x]=count;
					
							
						
				}	
			
			}*/
			
			

}

/*cluster-wise parallel computation of connectedness scores*/
__global__ void score(int*comm_conn_score,int*neigh_conn_score,int *c_neigh_list,edge_list *adj_list,cluster *cluster_list,int no_node,int KCORE){

    int x=threadIdx.x;
    
    int i,j,k;
    
    for(i=0;i<no_node;i++){
        if(c_neigh_list[x*no_node+i]>KCORE)
    		comm_conn_score[x*no_node+i]=(((c_neigh_list[x*no_node+i]-KCORE+1)*100)/(cluster_list[x].size-KCORE));
    	else
    	    comm_conn_score[x*no_node+i]=0; 
    	neigh_conn_score[x*no_node+i]=((c_neigh_list[x*no_node+i])*100)/adj_list[i*no_node].count;
    	
    }
    
    return;
}

/*Vertex-wise parallel computation of distribution of counts of scores...common kernel for community and neighbourhood connectedness scores*/
__global__ void bucket_fill(cluster*cluster_list,int *conn_score,int *bucket,int no_node,int maxi){

    int x=threadIdx.x;
    int i,k;
    
    
    
    for(i=0;i<no_node;i++)
    {
        
		for(k=1;k<20 && cluster_list[i].tag!=-1;k++)
		{
		        
				if(conn_score[threadIdx.x+i*no_node]>(k*5) && conn_score[threadIdx.x+i*no_node]<=(5*(k+1)))	
			{
					bucket[x*(maxi)+19-k]++;
		
			}	
		    
		
		}    
        if(cluster_list[i].tag!=-1)
        {
        
               				if(conn_score[threadIdx.x+i*no_node]>=0 && conn_score[threadIdx.x+no_node*i]<5)	
               				   bucket[x*(maxi)+19]++;
        
        }
    
    
    }    
	
    
    return;
}	

