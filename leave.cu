#include "hip/hip_runtime.h"
/*community-wise parallel computation for leave phase*/
__global__ void leave_phase(int *stay_listd,cluster *cluster_list,int *cluster_track_vertex,int *comm_conn_scored,int no_node,int *added_device,int KCORE,int *leaved)
{
            __shared__ int leave[1];
			int x=threadIdx.x,i,count=0;
			leave[0]=0;
			if(cluster_list[x].tag!=-1)
			{
			        
					for(i=0;i<no_node ;i++)
					{
					
							int y=added_device[x*no_node+i];
							if(y!=-1 && cluster_track_vertex[x*no_node+y]!=-1 && stay_listd[y]>comm_conn_scored[x*no_node+y])
							{
							          cluster_track_vertex[x*no_node+y]=-1;
							          cluster_list[x].size=cluster_list[x].size-1;
							    //      count++;
							          if(cluster_list[x].size<=KCORE)
							          {
							                cluster_list[x].tag=-1;
							               // leave[0]=0;
							          }
							          else
							          {
							                leave[0]=1;
							                
							            	__syncthreads();
							            	*leave=leave[0];
							            	
							           }
							           
							}
						//	if( y==-1 )
							
							//     count++; 
							    
							
					
					}
					//cluster_list[x].size=(no_node-count);
			  *leaved=leave[0];
			}
			
			
}
