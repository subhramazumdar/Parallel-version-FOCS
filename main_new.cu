#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <limits.h>
#include <float.h>
#include <malloc.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define byte 32
int KCORE = 2;
int OVL =50;
int buck;


#include "near_duplicate.h"
//#include "count.h"
#include "cluster_count.cu"
#include "counting.h"
#include "cut_off.cu"
#include "leave.cu"
#include "expand.cu"
/******************** CHANGE PARAMETER VALUES HERE ****************************/

//#define KCORE 3 //a node will have atleast KCORE connections (computation time is decreased with increase in value of KCORE beacuse of rapid filtering of nodes)
//#define OVL 0.6 //allow the amount(fraction) of overlap before the smaller of the communities is removed

/******************** CHANGE PARAMETER VALUES HERE ****************************/


/*Cuda kernel to find neighbours of nodes with degree>=KCORE...Also initializes the Added_i list..cluster_track_vertex is a list which stores the members of each cluster*/
__global__ void find_neighbour(cluster*cluster_list,int *cluster_track_vertex,edge_list*a_d,int *added,long int no_nodes,int*k)
{
//edge_list **a_d,
	int i;
	//cluster_list[threadIdx.x].tag=0;
	//if(cluster_list[threadIdx.x].tag==-1){
		//a_d[threadIdx.x]->count=0;
	if(a_d[(threadIdx.x)*no_nodes].count >=*k)
	  { 
	  
	     cluster_list[threadIdx.x].tag=1;
	    
		//
		
		cluster_list[threadIdx.x].head=threadIdx.x*no_nodes;
		for(i=0;i<no_nodes;i++)
		{
		    if(((threadIdx.x)*no_nodes+i)%no_nodes == 0)
		      cluster_track_vertex[(threadIdx.x)*no_nodes+a_d[(threadIdx.x)*no_nodes+i].vert]=1;
		      
		      
		      
		     if(((threadIdx.x)*no_nodes+i)%no_nodes!=0 && a_d[(threadIdx.x)*no_nodes+i].vert!=-1){
		      added[(threadIdx.x)*no_nodes+i]=a_d[(threadIdx.x)*no_nodes+i].vert;
		      (cluster_list[threadIdx.x].size)++;
		      cluster_track_vertex[(threadIdx.x)*no_nodes+a_d[(threadIdx.x)*no_nodes+i].vert]=1;
		}
		  
	}
  }
	
}
		

int main(int argc, char *argv[])
{

     hipEvent_t start, stop;
float time,time1=0;

	
	time_t start_time, end_time;

	int no_node,vert,vert1,maxi,*c_neigh_list,*c_neigh_listd,*comm_conn_scored,*neigh_conn_scored,*neigh_conn_score,*comm_conn_score,*stay_list,*stay_listd,*join_list,*join_listd,*now_added,*now_added_d;
    edge_list *adj_list,*temp,*adj_listd;	
    cluster*cluster_list,*cluster_listd;
	int i, j, vtx, adj, *temp2, mid, ll, ul, directed,flag=0,*flagd,*added,*added_device,*exapnd,*expand_d,count=0;
	
    
	char *syscmd, string[32];

	FILE *fp, *p;
    int *comm_vert_bucket,*comm_vert_bucketd,bucket_count=0;
    int *neigh_vert_bucket,*neigh_vert_bucketd,*leavd,*leave;
	int *cluster_track_vertex,*cluster_track_vertexd;
	leave=(int*)malloc(1*sizeof(int));
	exapnd=(int*)malloc(1*sizeof(int));
*leave=0,*exapnd=0;

	hipMalloc(&leavd,sizeof(int));
	hipMalloc(&expand_d,sizeof(int));

	
	if(argc < 2){
		printf("less input arguments.enter file name\n");
		exit(1);
	}
	else if(argc == 4){
		sscanf(argv[2], "%d", &KCORE);
		sscanf(argv[3], "%d", &OVL);
	}
	syscmd = (char *)malloc(200*sizeof(char));

	fp = fopen(argv[1],"r");
	fscanf(fp, "%d\t%d\n", &vtx, &adj);
	
	sprintf(string, "^%d\t%d$", adj, vtx);
	fclose(fp);
    OVL-=10;
	sprintf(syscmd, "grep -e '%s' %s| wc -l ", string, argv[1]);
	p = popen(syscmd, "r");
	fscanf(p,"%d", &directed);
	pclose(p);
	fprintf(stderr, "\ndirected %d", directed);

	
	if(directed == 0){
		sprintf(syscmd,"awk 'BEGIN{OFS=\"\t\";} {print $2,$1}' %s > temp", argv[1]);
		system(syscmd);
		sprintf(syscmd,"cat %s temp|sort -k 1,1n -k 2,2n > temp1", argv[1]);
		system(syscmd);
	}
	else{
		sprintf(syscmd,"sort -k 1,1n -k 2,2n %s > temp1", argv[1]);
		system(syscmd);
	}

	system("cut -f 1 temp1 > temp");
	sprintf(syscmd,"sort -n temp| uniq -c > %s.uniq", argv[1]);
	system(syscmd);
	system("rm temp");

	sprintf(syscmd, "wc -l %s.uniq", argv[1]);
	p = popen(syscmd, "r");
	fscanf(p,"%d", &no_node);
	pclose(p);
	fprintf(stderr, "\nNumber of nodes:\t%d\n", no_node);
    added=(int*)malloc(no_node*no_node*sizeof(int));//initial list of peripheral nodes added to in a cluster and then this modified using now_added list in expand phase	
    cluster_track_vertex=(int*)malloc(no_node*no_node*sizeof(int));//list of clusters to which each node belongs
    
    
  	adj_list=(edge_list*)malloc(no_node*no_node*sizeof(edge_list));//adjacency list representation of input graph
  	added=(int*)malloc(no_node*no_node*sizeof(int));
  	c_neigh_list=(int*)malloc(no_node*no_node*sizeof(int));//list no of neighbours of each vertex in each cluster 
  	for(i=0;i<no_node*no_node;i++)
  		c_neigh_list[i]=-1;
  	//hipMalloc(&adj_listd,no_node*sizeof(edge_list*));
  	cluster_list=(cluster*)malloc(no_node*sizeof(cluster));//stores all the clusters along with its members
  	stay_list=(int*)malloc(sizeof(int)*no_node);//list of stay cut-offs
  	join_list=(int*)malloc(sizeof(int)*no_node);//list of join cut-offs
  	now_added=(int*)malloc(no_node*no_node*sizeof(int));//to store list of peripheral nodes to be added in expand phase
  	printf("%d\n",no_node);
       // Allocate array on device
    hipMalloc( &cluster_listd, no_node*sizeof(cluster));
    hipMalloc(&cluster_track_vertexd,no_node*no_node*sizeof(int));
    
	hipMalloc( &flagd, sizeof(int));
	hipMalloc( &added_device,no_node*no_node*sizeof(int));
	hipMalloc( &stay_listd,no_node*sizeof(int));
	hipMalloc( &join_listd,no_node*sizeof(int));
	hipMalloc( &now_added_d,no_node*no_node*sizeof(int));
	
  	fp = fopen(argv[1],"r");
    //initializations
  	for(i=0;i<no_node*no_node;i++)
  	{

  			//hipMalloc( &(adj_listd[i]), sizeof(edge_list)); 
  			
  			adj_list[i].vert=-1;
  			adj_list[i].count=0;
  			added[i]=-1;
  			cluster_track_vertex[i]=-1;
  			now_added[i]=-1;

  			
  	
  	}
  	
  	for(i=0;i<no_node;i++){//{
        cluster_list[i].tag=-1;
        cluster_list[i].size=1;
    }
      
	maxi=0;
	for(i=0;i<no_node;i++)
  		  adj_list[i*no_node].vert=i;
  	while(fscanf(fp,"%d %d\n",&vert,&vert1) )
  	{
  		 
  		  //temp=(edge_list*)malloc(1*sizeof(edge_list));
  		/*  temp->vert=vert1;
  		  temp->next=NULL;
  		  adj_list[vert-1]->next=temp;*/
  		  adj_list[(vert-1)*no_node].count++;
  		//  printf("vert:%d,%d\t",vert,vert1);
  		
  		
  		
  		  
  		  
  		  for(i=0;i<no_node;i++)
  		  {
				if(adj_list[(vert-1)*no_node+i].vert==-1)
				{
  		  			adj_list[(vert-1)*no_node+i].vert=vert1-1;
  		  			printf("vert:%d,%d\t",vert,vert1);
  		  			break;	
  		  	    }
  		  		
  		  
  		  }
  		  if(feof(fp))
  		     break;
  		  
  		  if( adj_list[vert-1].count>maxi)
  		  		maxi= adj_list[vert-1].count;
  	}
  	if(maxi>=20)
  	{
  		comm_vert_bucket=(int*)malloc(maxi*no_node*sizeof(int));
  		neigh_vert_bucket=(int*)malloc(maxi*no_node*sizeof(int));
  		bucket_count=maxi;
  	}	
    else
    {		
  	    comm_vert_bucket=(int*)malloc(20*no_node*sizeof(int));
  	    neigh_vert_bucket=(int*)malloc(20*no_node*sizeof(int));
  	    bucket_count=20;
  	    
  	}    
  	
  	for(i=0;i<bucket_count*no_node;i++)
  	{
  	   comm_vert_bucket[i]=0;
  	   neigh_vert_bucket[i]=0;
  	}
  	hipMalloc(&comm_vert_bucketd,sizeof(int)*no_node*bucket_count);
  	hipMemcpy(comm_vert_bucketd,comm_vert_bucket,sizeof(int)*no_node*bucket_count,hipMemcpyHostToDevice);
  	hipMalloc(&neigh_vert_bucketd,sizeof(int)*no_node*bucket_count);
  	hipMemcpy(neigh_vert_bucketd,neigh_vert_bucket,sizeof(int)*no_node*bucket_count,hipMemcpyHostToDevice);
  	
  	printf("\nAdj list\n");
  	  for(i=0;i<no_node;i++){
    	for(j=0;j<no_node;j++){
    	
    		printf("%d:%d\t",i,adj_list[no_node*i+j].vert);	
    	
    	
    	}
    	printf("\n");
    
    }
  	size_t size = no_node * sizeof(edge_list);
    
    
    //for(i=0;i<no_node;i++)
     hipMalloc( &adj_listd, no_node*no_node*sizeof(edge_list));
    
  	
 // 	for(i=0;i<no_node;i++)
  	//	printf("%d\n",adj_list[i].count);
  	
  	fclose(fp);
  	
   	hipMemcpy(adj_listd, adj_list, no_node*no_node*sizeof(edge_list), hipMemcpyHostToDevice);
   /*	for(i=0;i<no_node;i++)
  	{
  	        	hipMemcpy(adj_listd[i], adj_list[i], sizeof(edge_list), hipMemcpyHostToDevice);
  	}*/
   	
   	
   	   	hipMemcpy(added_device, added, no_node*no_node*sizeof(int), hipMemcpyHostToDevice);
   	hipMemcpy(cluster_listd, cluster_list, no_node*sizeof(cluster), hipMemcpyHostToDevice);
   	hipMemcpy(flagd,&KCORE, sizeof(int), hipMemcpyHostToDevice);
    //start_time = time(NULL);
   	//adj_listd,
   	printf("syncing\n");
   	hipMemcpy(cluster_track_vertexd,cluster_track_vertex,no_node*no_node*sizeof(int),hipMemcpyHostToDevice);
   	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

   	find_neighbour <<< 1,no_node >>> (cluster_listd,cluster_track_vertexd,adj_listd,added_device,no_node,flagd);//call to cuda kernel for detecting neighbours of vertices which have degree >=KCORE
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time1, start, stop);
     time+=time1;
   	hipMemcpy(cluster_track_vertex,cluster_track_vertexd,no_node*no_node*sizeof(int),hipMemcpyDeviceToHost);
      	   	
   	
   	//hipMemcpy(&KCORE, flagd, sizeof(int), hipMemcpyDeviceToHost);
   	for(i=0;i<no_node;i++)
  		cluster_list[i].tag=0;
    
    hipMemcpy(added, added_device, no_node*no_node*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(cluster_list,cluster_listd, no_node*sizeof(cluster), hipMemcpyDeviceToHost);
    /*	printf("Cluster track vertex\n");
   	for(i=0;i<no_node;i++)
   	{
   	
   	     for(j=0;j<no_node && cluster_list[i].tag!=-1 ;j++)
   	        printf("%d\t",cluster_track_vertex[i*no_node+j]);
   	      
   	     printf("\n");   
  		//cluster_list[i].tag=0;
    
   	}
    printf("Cluster\n");
    for(i=0;i<no_node;i++)
    {
    		printf("cluster:%d\n",i+1);
    		if(cluster_list[i].tag!=-1)
    		{
    				int k=cluster_list[i].head;
    				for(int l=k;l<(k+no_node);l++)
    				{
    						printf("%d\t",adj_list[l].vert);
    				
    				}
    				printf("\n");
    			
    		}
    
    }
    printf("\n\n");*/
  /* 	for(i=0;i<no_node;i++)
  		adj_list[i*no_node].count=0;*/
  		
 
   	 hipMemcpy(adj_list, adj_listd, sizeof(edge_list)*no_node*no_node, hipMemcpyDeviceToHost);
   	 /*for(i=0;i<no_node;i++){
    	for(j=0;j<no_node;j++){
    	
    		  added[no_node*i+j]=0;	
    	
    	
    	}
    
    
    }*/
	 *exapnd=1;
	 	 	hipMemcpy(expand_d, exapnd, 1*sizeof(int), hipMemcpyHostToDevice);
	 while (*exapnd==1){

	 	*leave=1;
	 	for(i=0;i<no_node;i++)
	 	{
	 	
	 		stay_list[i]=0;
	 		join_list[i]=0;
	 	}

	 		 	hipMemcpy(leavd, leave, 1*sizeof(int), hipMemcpyHostToDevice);
	 	while (*leave==1){
	 	
	 		*leave=0;
	 		
	 	hipMemcpy(leavd, leave, 1*sizeof(int), hipMemcpyHostToDevice);
	 	hipMemcpy(added,added_device,sizeof(int)*no_node*no_node,hipMemcpyDeviceToHost);
	 
    //hipMemcpy(adj_list,adj_listd, size, hipMemcpyDeviceToHost);
    /*for(i=0;i<no_node;i++){
    	printf("tag=%d\n",cluster_list[i].tag);
    	printf("count=%d\n",adj_list[i].count);
    	
    	   
   // printf("flag=%d\n",*flagd);
    	//printf("tag=%d\n",adj_list[i]->count);
    }*/	
    
    /*printf("Added\n");
    for(i=0;i<no_node;i++){
    	for(j=0;j<no_node;j++){
    	
    		printf("%d\t",added[no_node*i+j]);	
    	
    	
    	}
    	printf("\n");
    
    }*/
	/*graph = (CGRAPH *)malloc(no_node *sizeof(CGRAPH));
	temp2 = (int *)malloc(no_node * sizeof(int));

	sprintf(syscmd, "%s.uniq", argv[1]);
	fp = fopen(syscmd,"r");*/
    printf("detecting duplicate community\n");
    hipMemcpy(cluster_track_vertexd,cluster_track_vertex,no_node*no_node*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(cluster_listd, cluster_list, no_node*sizeof(cluster), hipMemcpyHostToDevice);
    near_duplicate(cluster_list,no_node,OVL,cluster_track_vertex);//host function call to detect and delete near duplicate clusters
    hipMemcpy(cluster_track_vertex, cluster_track_vertexd, no_node*no_node*sizeof(int), hipMemcpyDeviceToHost); 
    
    hipMemcpy(adj_listd, adj_list, no_node*no_node*sizeof(edge_list), hipMemcpyHostToDevice);
   
    for(i=0;i<no_node;i++){
    	for(j=0;j<no_node;j++){
    	
    		c_neigh_list[no_node*i+j]=0;	
    	
    	
    	}
  
    
    }
    hipMalloc(&c_neigh_listd,no_node*no_node*sizeof(int));
    hipMemcpy(c_neigh_listd, c_neigh_list, no_node*no_node*sizeof(int), hipMemcpyHostToDevice);
    //compute_c_score<<<1,no_node>>>(cluster_listd,c_score_listd,no_node);
   // hipMemcpy(c_score_listd, c_score_list, no_node*no_node*sizeof(int), hipMemcpyDeviceToHost);
    hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    count_cluster <<< 1,no_node >>> (c_neigh_listd,adj_listd,cluster_listd,no_node);//call to cuda kernel to find the list of clusters to 		which a node belongs..c_neigh_list is a list which stores the list of clusters a node belongs to.
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time1, start, stop);
     time+=time1;

   // count_cluster(1,c_neigh_list,adj_list,cluster_list,no_node);
    hipMemcpy(c_neigh_list, c_neigh_listd, no_node*no_node*sizeof(int), hipMemcpyDeviceToHost);
    
    comm_conn_score=(int*)malloc(no_node*no_node*sizeof(int));
    neigh_conn_score=(int*)malloc(no_node*no_node*sizeof(int));
    
    hipMalloc(&comm_conn_scored,no_node*no_node*sizeof(int));
    hipMalloc(&neigh_conn_scored,no_node*no_node*sizeof(int));
    hipMemcpy(comm_conn_scored, comm_conn_score, no_node*no_node*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(neigh_conn_scored, neigh_conn_score, no_node*no_node*sizeof(int), hipMemcpyHostToDevice);
    hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    score<<<1,no_node>>>(comm_conn_scored,neigh_conn_scored,c_neigh_listd,adj_listd,cluster_listd,no_node,KCORE);//call to cuda kernel to 		compute connectedness scores
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time1, start, stop);
    time+=time1;
 	printf ("Time for the kernel, parallel shared: %f ms\n", time);
    hipMemcpy(comm_conn_score, comm_conn_scored, no_node*no_node*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(neigh_conn_score, neigh_conn_scored, no_node*no_node*sizeof(int), hipMemcpyDeviceToHost);
    for(i=0;i<no_node;i++){
      comm_conn_score[i]=0;
      neigh_conn_score[i]=0;
    }
    //for(i=0;i<no_node;i++)
    	//score(comm_conn_score,neigh_conn_score,c_neigh_list,adj_list,cluster_list,no_node,KCORE,i);
    /*for(i=0;i<no_node;i++){
    	for(j=0;j<no_node;j++){
    	
    		printf("%d\t",c_neigh_list[i*no_node+j]);	
    	
    	
    	}
    	printf("\n");
    
    }*/
  /*  printf("scores\n");
   
    for(i=0;i<no_node;i++){
    	for(j=0;j<no_node && cluster_list[i].tag!=-1;j++){
    	
    		printf("cluster : %d, Vertex : %d :: %d %d,%d,clu_size:%d\n",i+1,j+1,comm_conn_score[i*no_node+j],neigh_conn_score[i*no_node+j],adj_list[j*no_node].count,cluster_list[i].size);	
    	
    	
    	}
    	printf("\n");
    
    }*/
  /*  printf("tag\n");
 	for(i=0;i<no_node;i++){
 		printf("%d ",cluster_list[i].tag);
 	}
 	printf("\n");*/
     hipMemcpy(cluster_listd, cluster_list, no_node*sizeof(cluster), hipMemcpyHostToDevice);
     hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	bucket_fill<<<1,no_node>>>(cluster_listd,comm_conn_scored,comm_vert_bucketd,no_node,bucket_count);//to compute score distribution 		using community connectedness scores
	bucket_fill<<<1,no_node>>>(cluster_listd,neigh_conn_scored,neigh_vert_bucketd,no_node,bucket_count);//to compute score distribution 	using neighbourhood connectedness scores
   
     hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time1, start, stop);
     time+=time1;
      printf ("Time for the kernel, parallel shared: %f ms\n", time);
	 hipMemcpy(comm_vert_bucket, comm_vert_bucketd, bucket_count*no_node*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(neigh_vert_bucket, neigh_vert_bucketd, bucket_count*no_node*sizeof(int), hipMemcpyDeviceToHost);
/*	for(i=0;i<bucket_count*no_node;i++)
	{
	       if(comm_vert_bucket[i]!=0)   
			printf("i:%d,bucket count : %d,%d\t",i,comm_vert_bucket[i],neigh_vert_bucket[i]);
			
		    if(i!=0 && i%bucket_count==0)
			   printf("\n");
	}*/
	//hipMemcpy(comm_vert_bucketd,comm_vert_bucket,sizeof(int)*no_node*bucket_count,hipMemcpyHostToDevice);
	hipMemcpy(stay_listd,stay_list,no_node*sizeof(int), hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	compute_stay_cut_off<<<1,no_node>>>(stay_listd,comm_vert_bucketd,no_node,bucket_count);//cuda kernel to compute  stay cut-off
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time1, start, stop);
     time+=time1;

	hipMemcpy(stay_list, stay_listd, no_node*sizeof(int), hipMemcpyDeviceToHost);
	
	hipMemcpy(join_listd,join_list,no_node*sizeof(int), hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	compute_stay_cut_off<<<1,no_node>>>(join_listd,neigh_vert_bucketd,no_node,bucket_count);//cuda kernel to compute  join cut-off
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time1, start, stop);
     time+=time1;

	hipMemcpy(join_list, join_listd, no_node*sizeof(int), hipMemcpyDeviceToHost); 
	/*printf("\nstay\n");
	for(i=0;i<no_node;i++)
		printf("%d ",join_list[i]);
     	    printf("l=%d \n",*leave);	*/
     	    
    hipMemcpy(leavd, leave, 1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cluster_listd, cluster_list, no_node*sizeof(int), hipMemcpyHostToDevice); 
	hipMemcpy(cluster_track_vertexd, cluster_track_vertex, no_node*no_node*sizeof(int), hipMemcpyHostToDevice);  	    
	hipMemcpy(comm_conn_scored, comm_conn_score, no_node*no_node*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(neigh_conn_scored, neigh_conn_score, no_node*no_node*sizeof(int), hipMemcpyHostToDevice);
    hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	leave_phase<<<1,no_node>>>(stay_listd,cluster_listd,cluster_track_vertexd,comm_conn_scored,no_node,added_device,KCORE,leavd);//call to cuda kernel to delete communities
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time1, start, stop);
    time+=time1;
	hipMemcpy(leave, leavd, 1*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(cluster_list, cluster_listd, no_node*sizeof(int), hipMemcpyDeviceToHost); 
	hipMemcpy(cluster_track_vertex, cluster_track_vertexd, no_node*no_node*sizeof(int), hipMemcpyDeviceToHost); 
	 printf("Cluster\n");
    for(i=0;i<no_node;i++)
    {
    		//printf("cluster:%d,%d\n",i+1,cluster_list[i].size);
    		if(cluster_list[i].tag!=-1)
    		{
    		        count=0;
    				
    				for(j=0;j<no_node;j++)
    				{
    				  //printf("%d\t",cluster_track_vertex[i*no_node+j]);
    				  if(cluster_track_vertex[i*no_node+j]>0)
    				    count++; 
    				}  
    				if(count<=KCORE)
    				{
    				   cluster_list[i].tag=-1;
    				   cluster_list[i].size=count;
    				   
    				   continue;  
    				}   
    				/*printf("Added\n");
    				for(j=0;j<no_node;j++)
    				{
    				
    				if(added[i*no_node+j]!=-1)
    				  printf("%d\t",added[i*no_node+j]);
    				  
    				}  
    				printf("\n");  */
    			
    		}
    
    }
    //printf("l=%d \n",*leave);
    //exit(0);
    }
    
    *exapnd=0;
    
    hipMemcpy(now_added_d, now_added, no_node*no_node*sizeof(int), hipMemcpyHostToDevice); 	
    hipMemcpy(expand_d,exapnd,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(added_device, added, no_node*no_node*sizeof(int), hipMemcpyHostToDevice); 
    hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	//call to cuda kernel to expand communities
    expand_phase<<<1,no_node>>>(join_listd,cluster_listd,cluster_track_vertexd,neigh_conn_scored,no_node,added_device,now_added_d,adj_list,KCORE,expand_d,bucket_count-1);
   	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time1, start, stop);
     time+=time1;
      printf ("Time for the kernel, parallel shared: %f ms\n", time);
	hipMemcpy(added, now_added_d, no_node*no_node*sizeof(int), hipMemcpyDeviceToHost); 
	/*for(i=0;i<no_node;i++){
		for(j=0;j<no_node;j++){
		    if(added[j]!=-1)
				printf("i:%d j:%d ",i,added[j]);
		}
		printf("\n");
	}*/
	hipMemcpy(cluster_track_vertex, cluster_track_vertexd, no_node*no_node*sizeof(int), hipMemcpyDeviceToHost); 
//	hipMemcpy(leave,leavd,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(exapnd,expand_d,sizeof(int),hipMemcpyDeviceToHost);
	printf("%d %d\n",*leave,*exapnd);
	
	}
	
	 printf("\n\nCluster\n");
    for(i=0;i<no_node;i++)
    {
         if(cluster_list[i].tag!=-1)
         {
           
    		printf("cluster:%d,%d\n",i+1,cluster_list[i].tag);
    		for(j=0;j<no_node;j++)
    				{
    				  //printf("%d\t",cluster_track_vertex[i*no_node+j]);
    				  if(cluster_track_vertex[i*no_node+j]>0)
    				       printf("%d\t",j+1); 
    				}  
    				
    				printf("\n");		
          }
          
    }
 printf ("Time for the kernel, parallel shared: %f ms\n", time);
	hipFree(cluster_track_vertexd);
	hipFree(cluster_listd);
	hipFree(join_listd);
    hipFree(stay_listd);
    hipFree(comm_vert_bucketd);
    hipFree(neigh_vert_bucketd);
    hipFree(adj_listd);
	return 0;

}
